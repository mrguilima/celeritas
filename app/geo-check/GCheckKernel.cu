#include "hip/hip_runtime.h"
//---------------------------------*-Cudac-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file GCheckKernel.cu
//---------------------------------------------------------------------------//
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include <thrust/device_vector.h>

#include "GCheckKernel.hh"
#include "geometry/GeoInterface.hh"
#include "geometry/GeoTrackView.hh"
#include "geometry/LinearPropagator.hh"
#include "base/CollectionStateStore.hh"
//#include "base/OpaqueId.hh"
//#include "base/Array.hh"

using namespace celeritas;
using thrust::raw_pointer_cast;

namespace geo_check
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void gcheck_kernel(const GeoParamsCRefDevice  params,
                              const GeoStateRefDevice    state,
                              const GeoTrackInitializer* init,
                              int                        max_steps,
                              VolumeId*                  ids,
                              double*                    distances)
{
    CELER_EXPECT(params && state);
    CELER_EXPECT(max_steps > 0);

    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= state.size())
        return;

    celeritas::GeoTrackView     geo(params, state, tid);
    celeritas::LinearPropagator propagate(&geo);

    // Start track at the leftmost point in the requested direction
    geo = init[tid.get()];

    // Track along detector
    int istep = 0;
    do
    {
        // Save current ID and distance to travel
        auto step        = propagate();
        ids[istep]       = step.volume;
        distances[istep] = step.distance;
        printf("tid=%i step=%i: volid=%i, dist=%f\n",
               tid.get(),
               istep,
               (geo.is_outside() ? -1 : (int)step.volume.get()),
               step.distance);
        ++istep;
    } while (!geo.is_outside() && istep < max_steps);
}

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
/*!
 *  Run tracking on the GPU
 */
void run_gpu(GCheckInput input)
{
    // using StateStore
    //     = celeritas::CollectionStateStore<GeoStateData, MemSpace::device>;

    CELER_EXPECT(input.params);
    CELER_EXPECT(input.state);
    CELER_EXPECT(input.max_steps > 0);

    // Temporary device data for kernel
    // StateStore geo_states = StateStore(input.params.device_pointers(), 1);
    thrust::device_vector<GeoTrackInitializer> tracks(input.init.begin(),
                                                      input.init.end());
    thrust::device_vector<VolumeId> ids(input.init.size() * input.max_steps);
    thrust::device_vector<double>   distances(ids.size(), -1.0);

    // static const KernelParamCalculator calc_kernel_params(gcheck_kernel,
    // 							  "run_gpu");
    // auto launch_params = calc_launch_params(init.size());

    gcheck_kernel<<<1, 1>>>(input.params,
                            input.state,
                            raw_pointer_cast(input.init.data()),
                            input.max_steps,
                            raw_pointer_cast(ids.data()),
                            raw_pointer_cast(distances.data()));
    CELER_CUDA_CHECK_ERROR();

    CELER_CUDA_CALL(hipDeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace geo_check
