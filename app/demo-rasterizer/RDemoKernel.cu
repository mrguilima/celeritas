#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020-2022 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file demo-rasterizer/RDemoKernel.cu
//---------------------------------------------------------------------------//
#include "RDemoKernel.hh"

#include <cmath>

#include "corecel/Assert.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/geo/GeoMaterialView.hh"
#include "celeritas/geo/GeoTrackView.hh"
#include "celeritas/global/CoreParams.hh"

#include "ImageTrackView.hh"

using namespace celeritas;
using namespace demo_rasterizer;

namespace demo_rasterizer
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

using SPConstGeoMaterial = std::unique_ptr<celeritas::GeoMaterialParams>;

__device__ int geo_id(const GeoTrackView& geo, const GeoMaterialView& geomat)
{
    if (geo.is_outside())
        return -1;
    int mat_id = geomat.material_id(geo.volume_id()).get();
    return mat_id;
}

__global__ void trace_kernel(const GeoParamsCRefDevice geo_params,
                             const GeoMatCRefDevice    geomat_data,
                             const GeoStateRefDevice   geo_state,
                             const ImageData           image_state)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= image_state.dims[0])
        return;

    ImageTrackView image(image_state, tid);
    GeoTrackView   geo(geo_params, geo_state, tid);
    GeoMaterialView geomat(geomat_data);

    // Start track at the leftmost point in the requested direction
    geo = GeoTrackInitializer{image.start_pos(), image.start_dir()};

    int cur_id = geo_id(geo, geomat);

    // Track along each pixel
    for (unsigned int i = 0; i < image_state.dims[1]; ++i)
    {
        real_type pix_dist      = image_state.pixel_width;
        real_type max_dist      = 0;
        int       max_id        = cur_id;
        int       abort_counter = 32; // max number of crossings per pixel

        auto next = geo.find_next_step(pix_dist);
        while (next.boundary && pix_dist > 0)
        {
            CELER_ASSERT(next.distance <= pix_dist);
            // Move to geometry boundary
            pix_dist -= next.distance;

            if (max_id == cur_id)
            {
                max_dist += next.distance;
            }
            else if (next.distance > max_dist)
            {
                max_dist = next.distance;
                max_id   = cur_id;
            }

            // Cross surface and update post-crossing ID
            geo.move_to_boundary();
            geo.cross_boundary();
            cur_id = geo_id(geo, geomat);

            if (--abort_counter == 0)
            {
                // Reinitialize at end of pixel
                Real3 new_pos = image.start_pos();
                celeritas::axpy((i + 1) * image_state.pixel_width,
                                image.start_dir(),
                                &new_pos);
                geo      = GeoTrackInitializer{new_pos, image.start_dir()};
                pix_dist = 0;
            }
            if (pix_dist > 0)
            {
                // Next movement is to end of geo or pixel
                next = geo.find_next_step(pix_dist);
            }
        }

        if (pix_dist > 0)
        {
            // Move to pixel boundary
            geo.move_internal(pix_dist);
            if (pix_dist > max_dist)
            {
                max_dist = pix_dist;
                max_id   = cur_id;
            }
        }
        image.set_pixel(i, max_id);
    }
}
} // namespace

//---------------------------------------------------------------------------//
// KERNEL INTERFACE
//---------------------------------------------------------------------------//
void trace(const GeoParamsCRefDevice& geo_params,
           const GeoMatCRefDevice&    geomat_data,
           const GeoStateRefDevice&   geo_state,
           const ImageData&           image)
{
    CELER_EXPECT(image);

    CELER_LAUNCH_KERNEL(trace,
                        celeritas::device().default_block_size(),
                        image.dims[0],
                        geo_params,
                        geomat_data,
                        geo_state,
                        image);

    CELER_DEVICE_CALL_PREFIX(DeviceSynchronize());
}

//---------------------------------------------------------------------------//
} // namespace demo_rasterizer
